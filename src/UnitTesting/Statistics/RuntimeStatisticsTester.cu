#include "hip/hip_runtime.h"
#include "RuntimeStatisticsTester.cuh"

__device__ RuntimeStatisticsTester::RuntimeStatisticsTester(DataToDevice data)
{
    statistics = data.get_statistics_ptr();
}

__device__ bool RuntimeStatisticsTester::test_time(int64_t before,
        int64_t after, int64_t control)
{
    if (before > after) {
        printf("\tBefore (%lld) should not be greater than after (%lld)\n",
               before, after);
        return false;
    }

    if (before < 0) {
        printf("\tBefore (%lld) should not be negative\n", before);
        return false;
    }

    if (after < 0) {
        printf("\tAfter (%lld) should not be negative\n", after);
        return false;
    }


    if (before > LARGE_TIME) {
        printf("\tBefore (%lld) seems to large to be true\n", before);
        return false;
    }

    if (after > LARGE_TIME) {
        printf("\tAfter (%lld) seems to large to be true\n", after);
        return false;
    }

    if (after - before > control ) {
        printf("After (%lld) minus before (%lld)"
               " should not be greater than control (%lld)\n",
               after, before, control);
        return false;

    }

    return true;
}

__device__ bool RuntimeStatisticsTester::test_signal_job_start_stop()
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int initial_jobs = (statistics->get_n_jobs_run())[index];

    int jobs_to_add = 43;

    int64_t before = statistics->get_total_time_solving();

    int64_t control = - clock64();

    for (int i = 0; i < jobs_to_add; i++) {
        statistics->signal_job_start();
        statistics->signal_job_stop();
    }

    control += clock64();

    int64_t after = statistics->get_total_time_solving();

    int resulting_jobs = (statistics->get_n_jobs_run())[index];

    if (resulting_jobs != initial_jobs + jobs_to_add) {
        printf("\tNumber of jobs added (%d) added to the number of initial jobs (%d)"
               " and the number of jobs (%d) do not equal\n", jobs_to_add, initial_jobs,
               resulting_jobs);
        return false;
    }

    if (!test_time(before, after, control)) {
        return false;
    }

    return true;


}
__device__ bool RuntimeStatisticsTester::test_signal_decision_start_stop()
{
    int decision_to_make = 30;

    int64_t before = statistics->get_total_deciding_time();

    int64_t control = - clock64();

    for (int i = 0; i < decision_to_make; i++) {
        statistics->signal_decision_start();
        statistics->signal_decision_stop();
    }

    control += clock64();

    int64_t after = statistics->get_total_deciding_time();

    return test_time(before, after, control);
}

__device__ bool RuntimeStatisticsTester::test_signal_conflict_analysis_start_stop()
{
    int decision_to_make = 30;

    int64_t before = statistics->get_total_conflict_analyzing_time();

    int64_t control = - clock64();

    int before_decision_level = 3;
    int after_decision_level = 1;
    int total_decision_level = 0;

    for (int i = 0; i < decision_to_make; i++) {
        statistics->signal_conflict_analysis_start(before_decision_level);
        statistics->signal_conflict_analysis_stop(after_decision_level);

        total_decision_level += before_decision_level - after_decision_level;
    }

    int backtracked_levels = statistics->get_total_backtracked_levels();

    if (total_decision_level != backtracked_levels) {
        printf("\tNumber of backtracked levels (%d) is not what it should be (%d).\n", backtracked_levels,
               total_decision_level);
        return false;
    }

    control += clock64();

    int64_t after = statistics->get_total_conflict_analyzing_time();

    if (!test_time(before, after, control)) {
        return false;
    }

    return true;
}

__device__ bool RuntimeStatisticsTester::test_signal_backtrack_start_stop()
{
    int decision_to_make = 30;

    int64_t before = statistics->get_total_backtracking_time();

    int64_t control = - clock64();

    int decisions_before = 3;
    int decisions_after = 1;
    int total_decisions_before = statistics->get_sum_of_decision_before_backtracking();
    int total_decisions_after = statistics->get_sum_of_decision_after_backtracking();

    for (int i = 0; i < decision_to_make; i++) {
        statistics->signal_backtrack_start(decisions_before);
        statistics->signal_backtrack_stop(decisions_after);

        total_decisions_before += decisions_before;
        total_decisions_after += decisions_after;

    }

    int sum_of_decisions_before = statistics->get_sum_of_decision_before_backtracking();
    int sum_of_decisions_after = statistics->get_sum_of_decision_after_backtracking();

    if (sum_of_decisions_before != total_decisions_before) {
        printf("\tThe number of decisions done before backtracking (%d)"
               " does not match the number made (%d)\n", sum_of_decisions_before, total_decisions_before);
        return false;
    }

    if (sum_of_decisions_after != total_decisions_after) {
        printf("\tThe number of decisions done after backtracking (%d)"
               " does not match the number made (%d)\n", sum_of_decisions_after, total_decisions_after);
        return false;
    }

    control += clock64();

    int64_t after = statistics->get_total_backtracking_time();

    if (!test_time(before, after, control)) {
        return false;
    }

    return true;
}

__device__ bool RuntimeStatisticsTester::test_signal_start_stop()
{

    int index = threadIdx.x + blockIdx.x * blockDim.x;

#define TESTS_SIZES 10

    uint64_t starts[TESTS_SIZES] = {1, 21, 54, 76, 89, 123, 456, 786, 1023, 1990};
    uint64_t stops[TESTS_SIZES] = {6, 32, 65, 81, 97, 234, 512, 989, 1235, 2314};
    //int64_t total_sum_calc[gridDim.x*blockDim.x];// = 0;
    uint64_t *total_sum_calc = new uint64_t[gridDim.x * blockDim.x]; // = 0;
    uint64_t total_sum_oracle = 0;
    //int occurrences_calc[gridDim.x*blockDim.x];// = 0;
    int *occurrences_calc = new int[gridDim.x * blockDim.x]; // = 0;
    int occurrences_oracle = 0;

    total_sum_calc[index] = 0;
    occurrences_calc[index] = 0;


    for (int i = 0; i < TESTS_SIZES; i++) {
        total_sum_oracle += stops[i] - starts[i];
        occurrences_oracle++;

        statistics->signal_start(total_sum_calc, occurrences_calc, starts[i]);
        statistics->signal_stop(total_sum_calc, stops[i]);

        if (total_sum_calc[index] != total_sum_oracle) {
            printf("\tSum calculated (%lld) and expected sum (%lld) do not match.\n",
                   total_sum_calc, total_sum_oracle);
            return false;
        }

        if (occurrences_calc[index] != occurrences_oracle) {
            printf("\tOccurrences calculated (%d) and expected occurrences (%d) do not match.\n",
                   occurrences_calc, occurrences_oracle);
            return false;
        }

    }

    delete [] total_sum_calc;
    delete [] occurrences_calc;

    return true;
}

__device__ void RuntimeStatisticsTester::test_all()
{
    printf("Runtime statistics tester:\n");
    Tester::process_test(test_signal_start_stop(), "Test Signal Start/Stop");
    Tester::process_test(test_signal_job_start_stop(), "Test Signal Job Start/Stop");
    Tester::process_test(test_signal_decision_start_stop(), "Test Signal Decision Start/Stop");
    Tester::process_test(test_signal_conflict_analysis_start_stop(), "Test Signal Conflict Start/Stop");
    Tester::process_test(test_signal_backtrack_start_stop(), "Test Signal Backtrack Start/Stop");
}
