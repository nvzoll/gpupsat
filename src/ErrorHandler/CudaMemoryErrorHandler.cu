#include "CudaMemoryErrorHandler.cuh"

void check(hipError_t return_val, const char *message)
{
    if (return_val != hipSuccess) {
        printf("Error on %s, description: %s\n", message, hipGetErrorString(return_val));
        hipDeviceReset();
        exit(1);
    }
}
